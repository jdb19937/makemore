#include "hip/hip_runtime.h"
#define __MAKEMORE_MEGATRON_CU__ 1

#include <stdio.h>

#include <math.h>

#include <vector>
#include <map>

#include "cudamem.hh"
#include "megatron.hh"
#include "mapfile.hh"

namespace makemore {

double adam_b1 = 0.9;
double adam_b2 = 0.999;
double adam_b3 = 0.5;
double adam_eps = 1e-8;


__global__ void gpu_megatron_feed(
  const double *in,
  double *fin, double *out, double *fout,
  unsigned int inn, unsigned int outn,
  unsigned int wn,
  unsigned int **iwmap, unsigned int **owmap,
  unsigned int **iomap, unsigned int **oimap,
  unsigned int *wimap, unsigned int *womap,
  double *weight,
  double eta, double nu, bool activated,

  unsigned int inrn, unsigned int outrn, unsigned int mbn
) {
  unsigned int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;
  unsigned int outri = outi % outrn;
  unsigned int mbi = outi / outrn;

  unsigned int *inrip = oimap[outri];
  unsigned int *wip = owmap[outri];

  double sum = 0;
  while (*inrip) {
    unsigned int ini = mbi * inrn + *inrip - 1;
    unsigned int wi = *wip;

    sum += weight[wi] * in[ini];

    ++inrip;
    ++wip;
  }

  unsigned int wi = *wip;
  sum += weight[wi] * 1.0;

  if (activated) {
    double q = 1.0 / (1.0 + exp(-sum));
    out[outi] = q;
  } else {
    out[outi] = sum;
  }

  fout[outi] = 0.0;
} 

__global__ void gpu_megatron_train0(
  const double *in,
  double *fin, double *out, double *fout,
  unsigned int inn, unsigned int outn,
  unsigned int wn,
  unsigned int **iwmap, unsigned int **owmap,
  unsigned int **iomap, unsigned int **oimap,
  unsigned int *wimap, unsigned int *womap,
  double *weight,
  double eta, double nu, bool activated,

  unsigned int inrn, unsigned int outrn, unsigned int mbn
) {
  unsigned int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;

  double o = out[outi];
  double fo = fout[outi];

  if (o > 1.0)
    o = 1.0;
  else if (o < 0.0)
    o = 0.0;

  fout[outi] = fo * o * (1.0 - o);
}


__global__ void gpu_megatron_train1(
  const double *in,
  double *fin, double *out, double *fout,
  unsigned int inn, unsigned int outn,
  unsigned int wn,
  unsigned int **iwmap, unsigned int **owmap,
  unsigned int **iomap, unsigned int **oimap,
  unsigned int *wimap, unsigned int *womap,
  double *weight,
  double eta, double nu, bool activated,

  unsigned int inrn, unsigned int outrn, unsigned int mbn
) {
  unsigned int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  unsigned int inri = ini % inrn;
  unsigned int mbi = ini / inrn;

  unsigned int *outrip = iomap[inri];
  unsigned int *wip = iwmap[inri];

  double sum = 0;
  while (*outrip) {
    unsigned int outi = mbi * outrn + *outrip - 1;
    unsigned int wi = *wip;

    sum += weight[wi] * fout[outi];

    ++outrip;
    ++wip;
  }

  fin[ini] += sum;
}


__global__ void gpu_megatron_train2(
  const double *in,
  double *fin, double *out, double *fout,
  unsigned int inn, unsigned int outn,
  unsigned int wn,
  unsigned int **iwmap, unsigned int **owmap,
  unsigned int **iomap, unsigned int **oimap,
  unsigned int *wimap, unsigned int *womap,
  double *weight,

  double *m, double *v,  double a, double b1, double b2, double b3, double eps,

  bool activated,

  unsigned int inrn, unsigned int outrn, unsigned int mbn
) {
  unsigned int wi = blockIdx.x * blockDim.x + threadIdx.x;
  if (wi >= wn)
    return;

if (!(a > 0))
  return;


  unsigned int outri = womap[wi];
  --outri;
  unsigned int inri = wimap[wi];

  if (inri == 0) {
    for (unsigned int mbi = 0; mbi < mbn; ++mbi) {
      unsigned int outi = mbi * outrn + outri;
      double dw = fout[outi];
  m[wi] = b1 * m[wi] + (1 - b1) * dw;
  v[wi] = b2 * v[wi] + (1 - b2) * dw * dw;
  weight[wi] += a * m[wi] / (pow(v[wi], b3) + eps);
//weight[wi] += a * dw;
    }
  } else {
    --inri;

    for (unsigned int mbi = 0; mbi < mbn; ++mbi) {
      unsigned int outi = mbi * outrn + outri;
      unsigned int ini = mbi * inrn + inri;
      double dw = fout[outi] * in[ini];
  m[wi] = b1 * m[wi] + (1 - b1) * dw;
  v[wi] = b2 * v[wi] + (1 - b2) * dw * dw;
  weight[wi] += a * m[wi] / (pow(v[wi], b3) + eps);
//weight[wi] += a * dw;
    }
  }

}

const double *Megatron::feed(const double *_in, double *_fin) {
  in = _in;
  fin = _fin;

  int bs = 128;
  int gs = (outn + bs - 1) / bs;

  gpu_megatron_feed<<<gs, bs>>>(
    in, fin, out, fout, inn, outn,
    wn, iwmap, owmap, iomap, oimap, wimap, womap,
    weight, eta, 1.0, activated,
    inrn, outrn, mbn
  );

  return out;
}


void Megatron::train(double nu) {
  if (activated) {
    int bs0 = 128;
    int gs0 = (outn + bs0 - 1) / bs0;
    gpu_megatron_train0<<<gs0, bs0>>>(
      in, fin, out, fout, inn, outn,
      wn, iwmap, owmap, iomap, oimap, wimap, womap,
      weight, eta, nu, activated,
      inrn, outrn, mbn
    );
  }

  if (fin) {
    int bs1 = 128;
    int gs1 = (inn + bs1 - 1) / bs1;
    gpu_megatron_train1<<<gs1, bs1>>>(
      in, fin, out, fout, inn, outn,
      wn, iwmap, owmap, iomap, oimap, wimap, womap,
      weight, eta, nu, activated,
      inrn, outrn, mbn
    );
  }

  int bs2 = 128;
  int gs2 = (wn + bs2 - 1) / bs2;
  gpu_megatron_train2<<<gs2, bs2>>>(
    in, fin, out, fout, inn, outn,
    wn, iwmap, owmap, iomap, oimap, wimap, womap,
    weight,
m,v, eta*nu, adam_b1, adam_b2, adam_b3, adam_eps,
    activated,
    inrn, outrn, mbn
  );
}

Megatron::Megatron(const Wiring *_wire, Mapfile *_mapfile, unsigned int _mbn, double _eta, bool _activated)
  : Tron(_wire->inn * _mbn, _wire->outn * _mbn)
{
  mbn = _mbn;
  assert(mbn > 0);
  assert(inn % mbn == 0);
  inrn = inn / mbn;
  assert(outn % mbn == 0);
  outrn = outn / mbn;

  wire = _wire;
  mapfile = _mapfile;

  cumake(&out, outn);
  cumake(&fout, outn);

  cumake(&owmap, outrn);
  cumake(&oimap, outrn);
  cumake(&iomap, inrn);
  cumake(&iwmap, inrn);

  eta = _eta;
  activated = _activated;

  _makemaps();

  cumake(&weight, wn);
  mapfile->map(weight, wn);
  mapfile->load(weight);

  cumake(&m, wn);
  mapfile->map(m, wn);
  mapfile->load(m);

  cumake(&v, wn);
  mapfile->map(v, wn);
  mapfile->load(v);
}

Megatron::~Megatron() {
  cufree(out);
  cufree(fout);

  cufree(owmap);
  cufree(oimap);
  cufree(iwmap);
  cufree(iomap);
  cufree(womap);
  cufree(wimap);

  cufree(weight);
}

void Megatron::_makemaps() {
  using namespace std;

  vector< vector<unsigned int> > moi, mow, mio, miw;
  assert(wire->outn == outrn);
  assert(wire->inn == inrn);
  moi.resize(wire->outn);
  mow.resize(wire->outn);
  mio.resize(wire->inn);
  miw.resize(wire->inn);

  unsigned int wi = 0;
  wire->_makemaps(mio, miw, moi, mow);
  wi += wire->wn;

  for (unsigned int outri = 0; outri < wire->outn; ++outri) {
    moi[outri].push_back(0);
    mow[outri].push_back(wi);

    ++wi;
  }

  wn = wi;
  std::vector<unsigned int> mwi, mwo;
  mwi.resize(wn);
  mwo.resize(wn);

  for (unsigned int outri = 0; outri < wire->outn; ++outri) {
    for (auto wip = mow[outri].begin(); wip != mow[outri].end(); ++wip) {
      mwo[*wip] = outri + 1;
      mwi[*wip] = 0;
    }
  }
  for (unsigned int inri = 0; inri < wire->inn; ++inri) {
    for (auto wip = miw[inri].begin(); wip != miw[inri].end(); ++wip) {
      mwi[*wip] = inri + 1;
    }
  }

  for (unsigned int inri = 0; inri < wire->inn; ++inri) {
    mio[inri].push_back(0);
    miw[inri].push_back((unsigned int)-1);
  }

    
  unsigned int mapbufn = 0;
  for (unsigned int outri = 0; outri < outrn; ++outri) {
    const vector<unsigned int>& v = moi[outri];
    const vector<unsigned int>& w = mow[outri];
    assert(v.size());
    mapbufn += v.size();
    assert(w.size());
    mapbufn += w.size();
  }

  for (unsigned int inri = 0; inri < inrn; ++inri) {
    const vector<unsigned int>& v = mio[inri];
    const vector<unsigned int>& w = miw[inri];
    assert(v.size());
    mapbufn += v.size();
    assert(w.size());
    mapbufn += w.size();
  }

  cumake(&mapbuf, mapbufn);
  unsigned int mapbufi = 0;
  unsigned int *cmapbuf = new unsigned int[mapbufn];

  unsigned int **coimap = new unsigned int *[outrn];
  unsigned int **cowmap = new unsigned int *[outrn];
  unsigned int **ciomap = new unsigned int *[inrn];
  unsigned int **ciwmap = new unsigned int *[inrn];

  for (unsigned int outri = 0; outri < outrn; ++outri) {
    const vector<unsigned int>& v = moi[outri];
    const vector<unsigned int>& w = mow[outri];

    memcpy(cmapbuf + mapbufi, v.data(), v.size() * sizeof(unsigned int));
    coimap[outri] = mapbuf + mapbufi;
    mapbufi += v.size();

    memcpy(cmapbuf + mapbufi, w.data(), w.size() * sizeof(unsigned int));
    cowmap[outri] = mapbuf + mapbufi;
    mapbufi += w.size();
  }

  for (unsigned int inri = 0; inri < inrn; ++inri) {
    const vector<unsigned int>& v = mio[inri];
    const vector<unsigned int>& w = miw[inri];

    memcpy(cmapbuf + mapbufi, v.data(), v.size() * sizeof(unsigned int));
    ciomap[inri] = mapbuf + mapbufi;
    mapbufi += v.size();

    memcpy(cmapbuf + mapbufi, w.data(), w.size() * sizeof(unsigned int));
    ciwmap[inri] = mapbuf + mapbufi;
    mapbufi += w.size();
  }
  assert(mapbufi == mapbufn);

  encude(cmapbuf, mapbufn, mapbuf);
  delete[] cmapbuf; 

  encude(coimap, outrn, oimap);
  encude(cowmap, outrn, owmap);
  encude(ciomap, inrn, iomap);
  encude(ciwmap, inrn, iwmap);
  delete[] ciomap;
  delete[] ciwmap;
  delete[] coimap;
  delete[] cowmap;

  _mow = mow;

  cumake(&wimap, wn);
  encude(mwi.data(), wn, wimap);

  cumake(&womap, wn);
  encude(mwo.data(), wn, womap);
}

void Megatron::randomize(double disp) {
  using namespace std;

  double *cweight = new double[wn];

  for (unsigned int outri = 0; outri < outrn; ++outri) {
    const vector<unsigned int>& w = _mow[outri];
    assert(w.size());

    double iss = disp / sqrt(w.size() - 1);
    double sw = 0;
    for (unsigned int i = 0; i < w.size() - 1; ++i) {
      // double ww = iss * rnd(-1, 1);
      double ww = iss * randgauss();
      assert(w[i] < wn);
      cweight[w[i]] = ww;
      sw += ww;
    }
    assert(w[w.size() - 1] < wn);
//    cweight[w[w.size() - 1]] = 0;
    cweight[w[w.size() - 1]] = -sw/2.0;
  }

  encude(cweight, wn, weight);
  delete[] cweight;

  cuzero(m, wn);
  double *one = new double[wn];
  for (unsigned int wi = 0; wi < wn; ++wi)
    one[wi] = 1.0;
  encude(one, wn, v);
  delete[] one;
}

}
