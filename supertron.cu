#include "hip/hip_runtime.h"
#define __MAKEMORE_SUPERTRON_CU__ 1

#include <stdio.h>
#include <assert.h>

#include <math.h>

#include <vector>
#include <map>

#include "cudamem.hh"
#include "random.hh"
#include "supertron.hh"
#include "mapfile.hh"

namespace makemore {

inline __device__ int  shl(int a, int b) { return ((b < 0) ? (a >> -b) : (a << b)); }
inline            int dshl(int a, int b) { return ((b < 0) ? (a >> -b) : (a << b)); }

inline __device__ int  shr(int a, int b) { return ((b < 0) ? (a << -b) : (a >> b)); }
inline            int dshr(int a, int b) { return ((b < 0) ? (a << -b) : (a >> b)); }


inline __device__ bool dev_outi_to_ini_wi(const Supertron::Layer::Head* head, int outi, int i, int *inip, int *wip)
#include "outi_to_ini_wi.hh"
inline __device__ bool dev_ini_to_outi_wi(const Supertron::Layer::Head* head, int ini, int i, int *outip, int *wip)
#include "ini_to_outi_wi.hh"
inline __device__ bool dev_wi_to_ini_outi(const Supertron::Layer::Head* head, int wi, int i, int *inip, int *outip)
#include "wi_to_ini_outi.hh"

inline __device__ int dev_get_oiwtabn(const Supertron::Layer::Head* head)
#include "get_oiwtabn.hh"
inline __device__ int dev_get_iowtabn(const Supertron::Layer::Head* head)
#include "get_iowtabn.hh"
inline __device__ int dev_get_wiotabn(const Supertron::Layer::Head* head)
#include "get_wiotabn.hh"


#define shr dshr
#define shl dshl

inline bool outi_to_ini_wi(const Supertron::Layer::Head* head, int outi, int i, int *inip, int *wip)
#include "outi_to_ini_wi.hh"
inline bool ini_to_outi_wi(const Supertron::Layer::Head* head, int ini, int i, int *outip, int *wip)
#include "ini_to_outi_wi.hh"
inline bool wi_to_ini_outi(const Supertron::Layer::Head* head, int wi, int i, int *inip, int *outip)
#include "wi_to_ini_outi.hh"

inline int get_oiwtabn(const Supertron::Layer::Head* head)
#include "get_oiwtabn.hh"
inline int get_iowtabn(const Supertron::Layer::Head* head)
#include "get_iowtabn.hh"
inline int get_wiotabn(const Supertron::Layer::Head* head)
#include "get_wiotabn.hh"





__global__ void gpu_supertron_feed(
  Supertron::Layer layer
) {
  Supertron::Layer::Head *head = layer.head;
  int outn = head->outn;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;

  double *weight = layer.weight;
  const double *in = layer.in;

  double sum = 0;

  if (int *oiwtab = layer.oiwtab) {
    unsigned int oiwtabn = dev_get_oiwtabn(head);

    oiwtab += outi * oiwtabn * 2;
    for (int i = 0; i < oiwtabn; ++i, oiwtab += 2) {
      int ini = oiwtab[0];
      int wi = oiwtab[1];
      if (wi >= 0) {
        if (ini >= 0)
          sum += weight[wi] * in[ini];
        else if (ini == -1)
          sum += weight[wi];
      }
    }
  } else {
    int ini, wi;
    for (int i = 0; dev_outi_to_ini_wi(head, outi, i, &ini, &wi); ++i) {
      if (wi >= 0) {
        if (ini >= 0)
          sum += weight[wi] * in[ini];
        else if (ini == -1)
          sum += weight[wi];
      }
    }
  }

  if (head->activated) {
    double q = 1.0 / (1.0 + exp(-sum));
    layer.out[outi] = q;
  } else {
    layer.out[outi] = sum;
  }

  layer.fout[outi] = 0.0;
} 

__global__ void gpu_supertron_train0(
  Supertron::Layer layer, double a
) {
  Supertron::Layer::Head *head = layer.head;
  int outn = head->outn;
  int outi = blockIdx.x * blockDim.x + threadIdx.x;
  if (outi >= outn)
    return;
  if (!head->activated)
    return;

  double o = layer.out[outi];
  double fo = layer.fout[outi];

  if (o > 1.0)
    o = 1.0;
  else if (o < 0.0)
    o = 0.0;

  layer.fout[outi] = fo * o * (1.0 - o);
}




__global__ void gpu_supertron_train1(
  Supertron::Layer layer
) {
  Supertron::Layer::Head *head = layer.head;
  int inn = head->inn;
  unsigned int ini = blockIdx.x * blockDim.x + threadIdx.x;
  if (ini >= inn)
    return;

  double sum = 0;
  int outi, wi;

  double *weight = layer.weight;
  double *fout = layer.fout;

  if (int *iowtab = layer.iowtab) {
    unsigned int iowtabn = dev_get_iowtabn(head);

    iowtab += ini * iowtabn * 2;
    for (int i = 0; i < iowtabn; ++i, iowtab += 2) {
      outi = iowtab[0];
      wi = iowtab[1];

      if (outi >= 0 && wi >= 0)
        sum += weight[wi] * fout[outi];
    }
  } else {
    for (int i = 0; dev_ini_to_outi_wi(head, ini, i, &outi, &wi); ++i) {
      if (outi >= 0 && wi >= 0)
        sum += weight[wi] * fout[outi];
    }
  }

  layer.fin[ini] += sum;
}


__global__ void gpu_supertron_train2(
  Supertron::Layer layer, double a
) {
  Supertron::Layer::Head *head = layer.head;
  int wn = head->wn;

  unsigned int wi;
  int k;
  int i0, i1;
  unsigned int n = dev_get_wiotabn(head);

  if (layer.wbufk) {
    int q = blockIdx.x * blockDim.x + threadIdx.x;
    wi = q % wn;
    k = q / wn;
    if (k >= layer.wbufk)
      return;

    i0 = (k * n) / layer.wbufk;
    i1 = ((k + 1) * n) / layer.wbufk;
  } else {
    wi = blockIdx.x * blockDim.x + threadIdx.x;
    if (wi >= wn)
      return;
    k = -1;
    i0 = 0;
    i1 = n;
  }

  double *fout = layer.fout;
  const double *in = layer.in;

  double dw = 0;
  int nw = 0;


  if (int *wiotab = layer.wiotab) {
    unsigned int wiotabn = dev_get_wiotabn(head);

    wiotab += wi * wiotabn * 2 + i0 * 2;

    for (int i = i0; i < i1; ++i, wiotab += 2) {
      int ini = wiotab[0];
      int outi = wiotab[1];

      if (outi >= 0 && ini != -2) {
        dw += fout[outi] * (ini == -1 ? 1.0 : in[ini]);
        ++nw;
      }
    }

  } else {
    int ini, outi;

    for (int i = i0; i < i1; ++i) {
      dev_wi_to_ini_outi(head, wi, i, &ini, &outi);
      if (outi >= 0 && ini != -2) {
        dw += fout[outi] * (ini == -1 ? 1.0 : in[ini]);
        ++nw;
      }
    }
  }

  if (layer.wbufk) {
    layer.wbuf[wi * layer.wbufk + k] = dw;

    __syncthreads();

    if (k)
      return;

    double sdw = 0;
    for (k = 0; k < layer.wbufk; ++k)
      sdw += layer.wbuf[wi * layer.wbufk + k];
    dw = sdw;
  }

  if (nw > 1)
    dw /= (double)nw;

  if (head->adam) {
    layer.m[wi] = layer.head->adam_b1 * layer.m[wi] + (1.0 - layer.head->adam_b1) * dw;
    layer.v[wi] = layer.head->adam_b2 * layer.v[wi] + (1.0 - layer.head->adam_b2) * dw * dw;
    layer.weight[wi] += a * layer.m[wi] / (pow(layer.v[wi], layer.head->adam_b3) + layer.head->adam_eps);
  } else {
    layer.weight[wi] += a * dw;
  }
}

const double *Supertron::feed(const double *_in, double *_fin) {
  assert(layers.size() > 0);

  for (unsigned int li = 0; li < layers.size(); ++li) {
    Layer &lay = *layers[li];

    if (li > 0) {
      lay.in = layers[li - 1]->out;
      lay.fin = layers[li - 1]->fout;
    } else {
      lay.in = _in;
      lay.fin = _fin;
    }

    int bs = 256;
    int gs = (lay.outn + bs - 1) / bs;
//fprintf(stderr, "feeding layer %u\n", li);
    gpu_supertron_feed<<<gs, bs>>>(lay);
//fprintf(stderr, "    fed layer %u\n", li);

  }
  
  return layers[layers.size() - 1]->out;
}

void Supertron::target(const double *tgt) {
  assert(layers.size());
  Layer *lay = layers[layers.size() - 1];
  cusubvec(tgt, lay->out, lay->outn, lay->fout);
}


void Supertron::update_stats() {
  assert(layers.size());
  Layer *lay = layers[layers.size() - 1];

  double z = pow(1.0 - errdecay, (double)rounds);

  double nerr2 = sqrt(cusumsq(lay->fout, lay->outn) / (double)lay->outn);
  err2 *= (1.0 - z);
  err2 *= (1.0 - errdecay);
  err2 += errdecay * nerr2;
  err2 *= 1.0 / (1.0 - z * (1.0 - errdecay));

  double nerrm = cumaxabs(lay->fout, lay->outn);
  errm *= (1.0 - z);
  errm *= (1.0 - errdecay);
  errm += errdecay * nerrm;
  errm *= 1.0 / (1.0 - z * (1.0 - errdecay));

  ++rounds;
}

void Supertron::train(double nu) {
  unsigned int uli = 0;
  for (auto li = layers.rbegin(); li != layers.rend(); ++li) {
    Layer &lay = **li;

    if (lay.activated) {
      int bs0 = 256;
      int gs0 = (lay.outn + bs0 - 1) / bs0;
//fprintf(stderr, "training0 layer %u\n", uli);
      gpu_supertron_train0<<<gs0, bs0>>>(lay, nu);
//fprintf(stderr, "trained1  layer %u\n", uli);
    }

    if (lay.fin) {
      int bs1 = 256;
      int gs1 = (lay.inn + bs1 - 1) / bs1;
//fprintf(stderr, "training1 layer %u\n", uli);
      gpu_supertron_train1<<<gs1, bs1>>>(lay);
//fprintf(stderr, "trained1  layer %u\n", uli);
    }


    if (lay.wbufk) {
      int bs2 = 256;
      int gs2 = (lay.wn * lay.wbufk + bs2 - 1) / bs2;
      gpu_supertron_train2<<<gs2, bs2>>>(lay, nu);
    } else {
      int bs2 = 256;
      int gs2 = (lay.wn + bs2 - 1) / bs2;
//fprintf(stderr, "training2 layer %u\n", uli);
      gpu_supertron_train2<<<gs2, bs2>>>(lay, nu);
//fprintf(stderr, "trained2  layer %u\n", uli);
    }

    ++uli;
  }
}

Supertron::Supertron(Mapfile *_mapfile) {
  inn = 0;
  outn = 0;
  err2 = 0;
  errm = 0;
  rounds = 0;
  errdecay = 0.01;

  mapfile = _mapfile;

  cumake(&cunlayers, 1);
  mapfile->map(cunlayers, 1);
  mapfile->load(cunlayers);

  uint64_t nlayers;
  decude(cunlayers, 1, &nlayers);
  layers.resize(nlayers);

  for (unsigned int i = 0; i < nlayers; ++i) {
    Layer *lay = new Layer;
    cumake(&lay->head, 1);
    mapfile->map(lay->head, 1);
    mapfile->load(lay->head);

    Layer::Head head;
    decude(lay->head, 1, &head);
    lay->wn = head.wn;
    lay->inn = head.inn;
    lay->outn = head.outn;
    lay->activated = head.activated;

    cumake(&lay->out, head.outn);
    cumake(&lay->fout, head.outn);

    cumake(&lay->weight, head.wn);
    mapfile->map(lay->weight, head.wn);
    mapfile->load(lay->weight);

    cumake(&lay->m, head.wn);
    mapfile->map(lay->m, head.wn);
    mapfile->load(lay->m);

    cumake(&lay->v, head.wn);
    mapfile->map(lay->v, head.wn);
    mapfile->load(lay->v);

    lay->iowtab = NULL;
    lay->oiwtab = NULL;
    lay->wiotab = NULL;

    if (head.type == Supertron::Layer::TYPE_CONV) {
      lay->wbufk = 128;
      cumake(&lay->wbuf, lay->wn * lay->wbufk);
    }

#if 0
    {
      unsigned int oiwtabn = get_oiwtabn(&head);
      cumake(&lay->oiwtab, oiwtabn * head.outn * 2);
      mapfile->map(lay->oiwtab, oiwtabn * head.outn * 2);
      mapfile->load(lay->oiwtab);
    }

    {
      unsigned int iowtabn = get_iowtabn(&head);
      cumake(&lay->iowtab, iowtabn * head.inn * 2);
      mapfile->map(lay->iowtab, iowtabn * head.inn * 2);
      mapfile->load(lay->iowtab);
    }

    {
      unsigned int wiotabn = get_wiotabn(&head);
      cumake(&lay->wiotab, wiotabn * head.wn * 2);
      mapfile->map(lay->wiotab, wiotabn * head.wn * 2);
      mapfile->load(lay->wiotab);
    }
#endif


    layers[i] = lay;
  }

  if (nlayers) {
    inn = layers[0]->inn;
    outn = layers[layers.size() - 1]->outn;
  }
}

Supertron::~Supertron() {
  for (auto lay : layers) {
    cufree(lay->head);
    cufree(lay->out);
    cufree(lay->fout);
    cufree(lay->weight);
    cufree(lay->m);
    cufree(lay->v);
    delete lay;
  }
  if (cunlayers)
    cufree(cunlayers);
}

void Supertron::add_layer(
   Layer::Type type,
   unsigned int iw, unsigned int ih, unsigned int ic,
   unsigned int ow, unsigned int oh, unsigned int oc,
   unsigned int d, int s,
   bool activated
) {
  Layer::Head head;

  head.type = type;
  head.iw = iw;
  head.ih = ih;
  head.ic = ic;
  head.ow = ow;
  head.oh = oh;
  head.oc = oc;
  head.d = d;
  head.s = s;
  head.activated = activated;

  head.inn = iw * ih * ic;
  head.outn = ow * oh * oc;

  head.adam = true;
  head.adam_b1 = 0.9;
  head.adam_b2 = 0.999;
//  head.adam_b2 = 0.99;
  head.adam_b3 = 0.5;
  head.adam_eps = 1e-8;

  int d21 = (d * 2 + 1);
  int m = (d21 * d21 * ic + 1);
  int s2 = (s > 0) ? (1 << s) : 1;

  switch (head.type) {
  case Layer::TYPE_FULL:
    head.wn = (head.inn + 1) * head.outn;
    assert(d == 0);
    assert(s == 0);
    break;

  case Layer::TYPE_LOCAL:
    head.wn = head.outn * m;
fprintf(stderr, "%d, %d, %d, %d\n", ow, iw, s, (iw>>-s));
    assert(ow == shl(iw, s));
    assert(oh == shl(ih, s));
    break;

  case Layer::TYPE_CONV:
    head.wn = s2 * s2 * head.oc * m;
    assert(ow == shl(iw, s));
    assert(oh == shl(ih, s));
    break;

  case Layer::TYPE_YCONV:
    head.wn = s2 * oc * ow * m;
//fprintf(stderr, "s=%d iw=%u ow=%u\n", s, iw, ow);
    assert(ow == shl(iw, s));
    assert(oh == shl(ih, s));
    break;

  default:
    assert(0);
  }

fprintf(stderr, "wn=%d\n", head.wn);

  add_layer(head);
}

void Supertron::add_layer(const Supertron::Layer::Head &head) {
  uint64_t nlayers = layers.size();
  ++nlayers;
  encude(&nlayers, 1, cunlayers);

  Layer *lay = new Layer;
  cumake(&lay->head, 1);
  mapfile->map(lay->head, 1);
  encude(&head, 1, lay->head);

  cumake(&lay->out, head.outn);
  cumake(&lay->fout, head.outn);

  cumake(&lay->weight, head.wn);
  mapfile->map(lay->weight, head.wn);
  cumake(&lay->m, head.wn);
  mapfile->map(lay->m, head.wn);
  cumake(&lay->v, head.wn);
  mapfile->map(lay->v, head.wn);

  lay->wn = head.wn;
  lay->inn = head.inn;
  lay->outn = head.outn;
  lay->activated = head.activated;

  lay->iowtab = NULL;
  lay->oiwtab = NULL;
  lay->wiotab = NULL;

#if 0
  {
    unsigned int oiwtabn = get_oiwtabn(&head);
    cumake(&lay->oiwtab, oiwtabn * head.outn * 2);
    mapfile->map(lay->oiwtab, oiwtabn * head.outn * 2);

    int *oiwtab = new int[oiwtabn * head.outn * 2], *p = oiwtab;
    for (unsigned int outi = 0; outi < head.outn; ++outi) {
      for (int i = 0; i < oiwtabn; ++i, p += 2) {
        assert(outi_to_ini_wi(&head, outi, i, p + 0, p + 1));
      }

      int tmp0, tmp1;
      assert(!outi_to_ini_wi(&head, outi, oiwtabn, &tmp0, &tmp1));
    }
    encude(oiwtab, oiwtabn * head.outn * 2, lay->oiwtab);
    delete[] oiwtab;
  }

  {
    unsigned int iowtabn = get_iowtabn(&head);
    cumake(&lay->iowtab, iowtabn * head.inn * 2);
    mapfile->map(lay->iowtab, iowtabn * head.inn * 2);

    int *iowtab = new int[iowtabn * head.inn * 2], *p = iowtab;
    for (unsigned int ini = 0; ini < head.inn; ++ini) {
      for (int i = 0; i < iowtabn; ++i, p += 2) {
        assert(ini_to_outi_wi(&head, ini, i, p + 0, p + 1));
      }

      int tmp0, tmp1;
      assert(!ini_to_outi_wi(&head, ini, iowtabn, &tmp0, &tmp1));
    }
    encude(iowtab, iowtabn * head.inn * 2, lay->iowtab);
    delete[] iowtab;
  }

  {
    unsigned int wiotabn = get_wiotabn(&head);
    cumake(&lay->wiotab, wiotabn * head.wn * 2);
    mapfile->map(lay->wiotab, wiotabn * head.wn * 2);

    int *wiotab = new int[wiotabn * head.wn * 2], *p = wiotab;
    for (unsigned int wi = 0; wi < head.wn; ++wi) {
      for (int i = 0; i < wiotabn; ++i, p += 2) {
        assert(wi_to_ini_outi(&head, wi, i, p + 0, p + 1));
      }

      int tmp0, tmp1;
      assert(!wi_to_ini_outi(&head, wi, wiotabn, &tmp0, &tmp1));
    }
    encude(wiotab, wiotabn * head.wn * 2, lay->wiotab);
    delete[] wiotab;
  }
#endif

  layers.push_back(lay);

  if (layers.size() == 1)
    inn = lay->inn;
  outn = lay->outn;
}

void Supertron::randomize(double disp) {
  for (auto lay : layers) {
    Supertron::Layer::Head head;
    decude(lay->head, 1, &head);
    unsigned int s = get_oiwtabn(&head);

    double f = disp / sqrt((double)s + 1);

    double *weight = new double[lay->wn];
    for (unsigned int wi = 0; wi < lay->wn; ++wi)
      weight[wi] = randgauss() * f;
    encude(weight, lay->wn, lay->weight);
    delete[] weight;
  }
}

}

#if MAIN
using namespace makemore;

int main() {
  Mapfile mapfile("foos.map");
  Supertron s(&mapfile);

fprintf(stderr, "%lu\n", s.layers.size());
for (auto lay : s.layers) {
  Supertron::Layer::Head head;
  decude(lay->head, 1, &head);
  fprintf(stderr, "%u %u %u\n", head.inn, head.outn, head.wn);
}

#if 1
double *cuin;
cumake(&cuin, 256);
double *in = new double[256];
double *out = new double[256];

while (1) {
  for (int j = 0;j < 256; ++j) {
    in[j] = randrange(0, 1);
  }

  encude(in, 256, cuin);
  decude(s.feed(cuin), 256, out);

  s.target(cuin);
  s.update_stats();
  s.train(0.0001);

  if (s.rounds % 500 == 0) {
    for (unsigned int k = 0; k < 5; ++k)
      fprintf(stderr, "%lf\t", out[k]);
    fprintf(stderr, "\n");
    fprintf(stderr, "err2=%lf\n", s.err2);

    s.mapfile->save();
  }
}

#else
  s.add_layer(Supertron::Layer::TYPE_YCONV,
    16, 16, 1,
    32, 32, 1,
    1, 1, true
  );

  s.add_layer(Supertron::Layer::TYPE_YCONV,
    32, 32, 1,
    16, 16, 1,
    1, -1, false
  );
    
  s.randomize(1.0);
  s.mapfile->save();
#endif
  return 0;
}
#endif
