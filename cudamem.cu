#include "hip/hip_runtime.h"
#define __MAKEMORE_CUDAMEM_CU__ 1
#include "cudamem.hh"

#include <assert.h>
#include <stdio.h>

void encudev(const void *a, unsigned int n, void *da) {
  hipMemcpy(da, a, n, hipMemcpyHostToDevice);
}

void decudev(const void *da, unsigned int n, void *a) {
  hipMemcpy(a, da, n, hipMemcpyDeviceToHost);
}

void cumakev(void **dp, unsigned int n) {
  void *d = NULL;
  assert(0 == hipMalloc((void **)&d, n));
  assert(d != NULL);
  assert(dp != NULL);
  *dp = d;
}

void cufreev(void *x) {
  hipFree(x);
}

void cuzerov(void *x, unsigned int n) {
  hipMemset((void *)x, 0, n);
}

void cucopyv(const void *x, unsigned int n, void *y) {
  hipMemcpy(y, x, n, hipMemcpyDeviceToDevice);
}


__global__ void gpu_cuaddvec(const double *a, const double *b, double *c, int n) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
      c[i] = a[i] + b[i];
}

void cuaddvec(const double *a, const double *b, unsigned int n, double *c) {
  int bs = 128;
  int gs = ((n + bs - 1) / bs);
  gpu_cuaddvec<<<gs, bs>>>(a, b, c, n);
}
